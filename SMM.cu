
#include <hip/hip_runtime.h>
#include <time.h>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// #define NUM_THREADS_PER_BLOCK 	256
// #define NUM_BLOCKS 		16
#define PRINT_TIME 		1
#define SM_ARR_LEN		2048
#define TOL			5e-2
#define GIG                     1000000000
// #define CPG                     3.07
// #define IMUL(a, b) __mul24(a, b)
#define BLOCK_SIZE 128
// #define TILE_WIDTH 16
#define SPARSITY 0.05
#define FULL_MASK 0xffffffff

typedef float data_t;

void initializeArray1D(float *arr, int len, float seed);
void initializeSparseMatrixCSR(int *row_offset, int len, int *col_indices, float *values, float seed);

//Md - matrix  
//Nd - vector
//y - result 
// __global__ void MMK(float* Md, float* Nd, float* y)
// {
//     int col = blockDim.x * blockIdx.x + threadIdx.x;
//     int row = blockDim.y * blockIdx.y + threadIdx.y;
//     int num_row = SM_ARR_LEN / BLOCK_SIZE;
//     int k, i;
//     float sum = 0.0f;
//     for (i = 0; i < num_row; i++) {
//         if (col < SM_ARR_LEN || row < SM_ARR_LEN) {
//             for(k = 0; k < BLOCK_SIZE; k++){
//                 sum += Md[row * BLOCK_SIZE + k] * Nd[k];
//             }
//             y[i] = sum;  
//         }
//     }
// }

__device__ float warp_reduce(float val) 
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2) 
        val += __shfl_down_sync (FULL_MASK, val, offset);
    return val;
}

// __global__ void SpMV(int *row_off, float *val, int *col, float *y, float *x)
// {
//     int row = blockDim.y * blockIdx.y + threadIdx.y;
//     int numOfRows = SM_ARR_LEN / BLOCK_SIZE;
//     int i, j;
//     float sum = 0.0f;                                           

//     for (i=0; i < numOfRows; ++i) {
//         if (row < numOfRows) {
//             y[row] = 0.0;
//             for (j=row_off[row]; j<row_off[row+1]; ++j)
//                 sum += val[j] * x[col[j]];
//             y[row] = sum;
//         }   
//     }
// }

__global__ void edge_softmax_forward(int *row_off, float *val, float *y)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int numOfRows = SM_ARR_LEN / BLOCK_SIZE;
    int i, j, k, l;
    float max_score, exp_value, sum;                                         

    for (i=0; i < numOfRows; ++i) {
        if (row < numOfRows) {
            y[row] = 0.0;
            //find max edge value
            for (j=row_off[row]; j<row_off[row+1]; ++j){
                max_score = max(max_score, val[j]);
            }
            //update edge value && find sum of exp
            for (k=row_off[row]; k<row_off[row+1]; ++k) {
                val[k] = val[k] - max_score;
                exp_value = exp(val[k]);
                sum += exp_value;
            }
            
            for (l=row_off[row]; l<row_off[row+1]; ++l) {
                y[row] = exp_value / sum;
            }
        }   
    }
}

__global__ void edge_softmax_forward_warp(int *row_off, float *val, float *y)
{
    int threadId = blockDim.y * blockIdx.y + threadIdx.y;
    int numOfRows = SM_ARR_LEN / BLOCK_SIZE;
    int i, j, k, l;
    float max_score; 
    float exp_value; 
    float sum, res;  
    int warp_id = threadId / 32;
    int lane = threadId % 32; 
    int row = warp_id; //one warp per row

    for (i=0; i < numOfRows; ++i) {
        if (row < numOfRows) {
            y[row] = 0.0;
            for (j=row_off[row]; j<row_off[row+1]; ++j){
                max_score = max(max_score, val[j]);
            }
               
            for (k=row_off[row]; k<row_off[row+1]; ++k) {
                val[k] = val[k] - max_score;
                exp_value = exp(val[k]);
                sum += exp_value;
            }

            //inter communication in warp
            res = warp_reduce(sum);

            if (lane == 0 && row < numOfRows) {
                sum = res;
            }
                
            for (l=row_off[row]; l<row_off[row+1]; ++l) {
                y[row] = exp_value / sum;
            }
        }   
        
    }
}


int compare(float* h_result, float* h_result_gold){
    int i;
    int errCount =0;
    int zeroCount = 0;
    for(i = 0; i < SM_ARR_LEN*SM_ARR_LEN; i++) {
        if (abs(h_result_gold[i] - h_result[i]) > TOL*h_result_gold[i]) {
            errCount++;
        }
        if(h_result[i]==0)
            zeroCount++;
    }
    if (zeroCount>0)
        errCount = -1;
    return errCount;
}

float errorCal(float* h_result, float* h_result_gold){
    int i;
    float error = 0;
    for(i = 0; i < SM_ARR_LEN*SM_ARR_LEN; i++) {
        if(abs(h_result_gold[i] - h_result[i])>error)
            error =  abs(h_result_gold[i] - h_result[i]);
    }
    return error;
}

double interval(struct timespec start, struct timespec end)
{
    struct timespec temp;
    temp.tv_sec = end.tv_sec - start.tv_sec;
    temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    if (temp.tv_nsec < 0) {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }
    return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

int main(int argc, char **argv){
    int arrLen = 0;

    // GPU Timing variables
    hipEvent_t start, stop, start2, stop2;
    float elapsed_gpu;

    // Arrays on GPU global memoryc
    //Md - matrix, Nd - vector, y - result matrix
    float *Md;
    float *Nd;
    float *y;
    int *row_offset;
    int *col_indices;
    float *value;
    float *x;

    // Arrays on the host memory
    float *Md_h;
    float *y_h;
    float *Nd_h;

    if (argc > 1) {
        arrLen  = atoi(argv[1]);
    }
    else {
        arrLen = SM_ARR_LEN * SM_ARR_LEN;
    }


    printf("Length of the array = %d\n", arrLen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

    // Allocate GPU memory
    size_t allocSize = arrLen * sizeof(float);
    size_t vectorSize = SM_ARR_LEN * sizeof(float);
    size_t allocSize_int = (SM_ARR_LEN * SM_ARR_LEN) * sizeof(int);
    size_t row_offset_size = (SM_ARR_LEN + 1) * sizeof(int);
    // CUDA_SAFE_CALL(cudaMalloc((void **)&Md, allocSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&y, vectorSize));
    // CUDA_SAFE_CALL(cudaMalloc((void **)&Nd, vectorSize));
    CUDA_SAFE_CALL(hipMalloc((void **)&row_offset, row_offset_size));
    // CUDA_SAFE_CALL(cudaMalloc((void **)&col_indices, allocSize_int));
    CUDA_SAFE_CALL(hipMalloc((void **)&value, allocSize));
    // CUDA_SAFE_CALL(cudaMalloc((void **)&x, allocSize));

    // Allocate arrays on host memory
    y_h		           = (float *) malloc(allocSize);
    // Md_h		           = (float *) malloc(allocSize);
    // Nd_h		           = (float *) malloc(vectorSize);
    int *row_offset_h = (int *)malloc(row_offset_size);
    int *col_indices_h = (int *)malloc(allocSize_int);
    float *values_h = (float *)malloc(allocSize);
    // float *x_h = (float *)malloc(allocSize);


    // Initialize the host arrays
    printf("\nInitializing the arrays ...");
    // Arrays are initialized with a known seed for reproducability
    // initializeArray1D(Md_h, arrLen, 0.53);
    //vector 
    // initializeArray1D(x_h, SM_ARR_LEN, 0.54);
    //sparse matrix
    initializeSparseMatrixCSR(row_offset_h, SM_ARR_LEN, col_indices_h, values_h, 0.54);
    printf("\t... done\n\n");


#if PRINT_TIME
    // Create the cuda events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record event on the default stream
    hipEventRecord(start, 0);
#endif

    // Transfer the arrays to the GPU memory
    // CUDA_SAFE_CALL(cudaMemcpy(Md, Md_h, allocSize, cudaMemcpyHostToDevice));
    // CUDA_SAFE_CALL(cudaMemcpy(Nd, Nd_h, vectorSize, cudaMemcpyHostToDevice));

    CUDA_SAFE_CALL(hipMemcpy(row_offset, row_offset_h, row_offset_size, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(value, values_h, allocSize, hipMemcpyHostToDevice));
    // CUDA_SAFE_CALL(cudaMemcpy(col_indices, col_indices_h, allocSize_int, cudaMemcpyHostToDevice));
    // CUDA_SAFE_CALL(cudaMemcpy(x, x_h, allocSize, cudaMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(y, y_h, allocSize, hipMemcpyHostToDevice));


    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);

    dim3 dimGrid(SM_ARR_LEN, SM_ARR_LEN);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // Launch the kernel
    edge_softmax_forward_warp<<<dimGrid, dimBlock>>>(row_offset, value, y);

    // timer for kernel execution
    hipEventRecord(stop2,0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsed_gpu, start2, stop2);
    printf("\nGPU kernel execution time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    // Check for errors during launch
    CUDA_SAFE_CALL(hipPeekAtLastError());

    // Transfer the results back to the host
    CUDA_SAFE_CALL(hipMemcpy(y_h, y, allocSize, hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(cudaMemcpy(col_indices_h, col_indices, allocSize_int, cudaMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(row_offset_h, row_offset, row_offset_size, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(values_h, value, allocSize, hipMemcpyDeviceToHost));
    // CUDA_SAFE_CALL(cudaMemcpy(x_h, x, allocSize, cudaMemcpyDeviceToHost));

#if PRINT_TIME
    // Stop and destroy the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_gpu, start, stop);
    printf("\nGPU start-to-finish time: %f (msec)\n", elapsed_gpu);
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif

    // printf("\nCompare: %d\n\n\n",compare(y_h,y_h_gold));
    // printf("\nBiggest Error: %f\n\n\n",errorCal(y_h,y_h_gold));

    // Free-up device and host memory
    CUDA_SAFE_CALL(hipFree(y));
    CUDA_SAFE_CALL(hipFree(value));
    // CUDA_SAFE_CALL(cudaFree(x));
    // CUDA_SAFE_CALL(cudaFree(col_indices));
    CUDA_SAFE_CALL(hipFree(row_offset));


    free(y_h);
    free(values_h);
    // free(col_indices_h);
    free(row_offset_h);
    // free(x_h);

    return 0;
}

struct timespec diff(struct timespec start, struct timespec end)
{
    struct timespec temp;
    if ((end.tv_nsec-start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
    } else {
        temp.tv_sec = end.tv_sec-start.tv_sec;
        temp.tv_nsec = end.tv_nsec-start.tv_nsec;
    }
    return temp;
}


void initializeArray1D(float *arr, int len, float seed) {
    int i;
    float randNum;
    srand(seed);

    for (i = 0; i < len; i++) {
        randNum = (float) (rand() / 100000);
        arr[i] = randNum;
    }
}

void initializeSparseMatrixCSR(int *row_offset, int len, int *col_indices, float *values, float seed) {
    //num of non-zero elements
    int nnz = 0;
    int i, j;
   
    srand(seed);

    for (i = 0; i < len; ++i) {
        // row_offset[i] = row_offset[i - 1];
        for (int j = 0; j < len; ++j) {
            if ((float)rand() / RAND_MAX < SPARSITY) {
                col_indices[nnz] = j;
                values[nnz] = (float)rand() / RAND_MAX;
                nnz++;
                // row_offset[nnz]++;
            }
        }
        row_offset[i + 1] = nnz;
    }
}

